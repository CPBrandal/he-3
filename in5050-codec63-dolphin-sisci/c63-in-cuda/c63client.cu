#include "hip/hip_runtime.h"
#include <assert.h>
#include <errno.h>
#include <getopt.h>
#include <limits.h>
#include <math.h>
#include <stdint.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#include <sisci_error.h>
#include <sisci_api.h>

#include "c63.h"
#include "c63_write.h"
#include "common.h"
#include "tables.h"

static char *output_file, *input_file;
FILE *outfile;

static uint32_t remote_node = 0;
static int limit_numframes = 0;

static uint32_t width;
static uint32_t height;

/* getopt */
extern int optind;
extern char *optarg;

/* Read planar YUV frames with 4:2:0 chroma sub-sampling */
static yuv_t *read_yuv(FILE *file, struct c63_common *cm)
{
    size_t len = 0;
    yuv_t *image = (yuv_t *)malloc(sizeof(*image));

    /* Read Y. The size of Y is the same as the size of the image. */
    image->Y = (uint8_t *)calloc(1, cm->padw[Y_COMPONENT] * cm->padh[Y_COMPONENT]);
    len += fread(image->Y, 1, width * height, file);

    /* Read U. Given 4:2:0 chroma sub-sampling, the size is 1/4 of Y */
    image->U = (uint8_t *)calloc(1, cm->padw[U_COMPONENT] * cm->padh[U_COMPONENT]);
    len += fread(image->U, 1, (width * height) / 4, file);

    /* Read V. Given 4:2:0 chroma sub-sampling, the size is 1/4 of Y. */
    image->V = (uint8_t *)calloc(1, cm->padw[V_COMPONENT] * cm->padh[V_COMPONENT]);
    len += fread(image->V, 1, (width * height) / 4, file);

    if (ferror(file))
    {
        perror("ferror");
        exit(EXIT_FAILURE);
    }

    if (feof(file))
    {
        free(image->Y);
        free(image->U);
        free(image->V);
        free(image);
        return NULL;
    }
    else if (len != width * height * 1.5)
    {
        fprintf(stderr, "Reached end of file, but incorrect bytes read.\n");
        fprintf(stderr, "Wrong input? (height: %d width: %d)\n", height, width);
        free(image->Y);
        free(image->U);
        free(image->V);
        free(image);
        return NULL;
    }

    return image;
}

struct c63_common *
init_c63_enc( int width, int height )
{
    int i;

    /* calloc() sets allocated memory to zero */
    c63_common *cm =
        ( c63_common * ) calloc( 1, sizeof( struct c63_common ) );

    cm->width = width;
    cm->height = height;

    cm->padw[Y_COMPONENT] = cm->ypw =
        ( uint32_t ) ( ceil( width / 16.0f ) * 16 );
    cm->padh[Y_COMPONENT] = cm->yph =
        ( uint32_t ) ( ceil( height / 16.0f ) * 16 );
    cm->padw[U_COMPONENT] = cm->upw =
        ( uint32_t ) ( ceil( width * UX / ( YX * 8.0f ) ) * 8 );
    cm->padh[U_COMPONENT] = cm->uph =
        ( uint32_t ) ( ceil( height * UY / ( YY * 8.0f ) ) * 8 );
    cm->padw[V_COMPONENT] = cm->vpw =
        ( uint32_t ) ( ceil( width * VX / ( YX * 8.0f ) ) * 8 );
    cm->padh[V_COMPONENT] = cm->vph =
        ( uint32_t ) ( ceil( height * VY / ( YY * 8.0f ) ) * 8 );

    cm->mb_cols = cm->ypw / 8;
    cm->mb_rows = cm->yph / 8;

    /* Quality parameters -- Home exam deliveries should have original values,
       i.e., quantization factor should be 25, search range should be 16, and the
       keyframe interval should be 100. */
    cm->qp = 25;                // Constant quantization factor. Range: [1..50]
    cm->me_search_range = 16;   // Pixels in every direction
    cm->keyframe_interval = 100;        // Distance between keyframes

    /* Initialize quantization tables */
    for ( i = 0; i < 64; ++i )
    {
        cm->quanttbl[Y_COMPONENT][i] = yquanttbl_def[i] / ( cm->qp / 10.0 );
        cm->quanttbl[U_COMPONENT][i] = uvquanttbl_def[i] / ( cm->qp / 10.0 );
        cm->quanttbl[V_COMPONENT][i] = uvquanttbl_def[i] / ( cm->qp / 10.0 );
    }

    return cm;
}

void
free_c63_enc( struct c63_common *cm )
{
    destroy_frame( cm->curframe );
    free( cm );
}

/* Main client processing loop */
int main_client_loop(struct c63_common *cm, FILE *infile, int limit_numframes,
    volatile struct client_segment *local_seg,
    volatile struct server_segment *remote_seg,
    sci_dma_queue_t dma_queue,
    sci_local_segment_t local_segment,
    sci_remote_segment_t remote_segment) 
{
yuv_t *image;
int numframes = 0;
sci_error_t error;

printf("Client: Sending dimensions (width=%u, height=%u)\n", width, height);

// Prepare the dimensions data
struct dimensions_data dim_data;
dim_data.width = width;
dim_data.height = height;

// Copy dimensions to the message buffer
memcpy((void*)local_seg->message_buffer, &dim_data, sizeof(struct dimensions_data));
local_seg->packet.data_size = sizeof(struct dimensions_data);

// Use DMA to transfer the dimensions
SCIStartDmaTransfer(dma_queue, 
       local_segment,
       remote_segment,
       offsetof(struct client_segment, message_buffer),
       sizeof(struct dimensions_data),
       offsetof(struct server_segment, message_buffer),
       NO_CALLBACK,
       NULL,
       NO_FLAGS,
       &error);

if (error != SCI_ERR_OK) {
fprintf(stderr, "Client: SCIStartDmaTransfer for dimensions failed - Error code 0x%x\n", error);
return -1;
}

// Wait for DMA transfer to complete
SCIWaitForDMAQueue(dma_queue, SCI_INFINITE_TIMEOUT, NO_FLAGS, &error);
if (error != SCI_ERR_OK) {
fprintf(stderr, "Client: SCIWaitForDMAQueue for dimensions failed - Error code 0x%x\n", error);
return -1;
}

// Signal server that dimensions are ready
SCIFlush(NULL, NO_FLAGS);
remote_seg->packet.cmd = CMD_DIMENSIONS;
SCIFlush(NULL, NO_FLAGS);

// Wait for server acknowledgment
local_seg->packet.cmd = CMD_INVALID;
while (local_seg->packet.cmd != CMD_DIMENSIONS_ACK) {
// Just wait
}

// Verify echoed dimensions
struct dimensions_data received_dim;
memcpy(&received_dim, (void*)local_seg->message_buffer, sizeof(struct dimensions_data));

if (received_dim.width != width || received_dim.height != height) {
fprintf(stderr, "Client: Server responded with incorrect dimensions (width=%u, height=%u)\n", 
received_dim.width, received_dim.height);
return -1;
}

printf("Client: Dimensions verified, starting video encoding\n");

while (1) {
// Read YUV frame
image = read_yuv(infile, cm);
if (!image) break;

printf("Processing frame %d, ", numframes);

// Send frame number to server
*((int*)local_seg->message_buffer) = numframes;
local_seg->packet.data_size = sizeof(int);

// Use DMA to transfer the frame number
SCIStartDmaTransfer(dma_queue, 
           local_segment,
           remote_segment,
           offsetof(struct client_segment, message_buffer),
           local_seg->packet.data_size,
           offsetof(struct server_segment, message_buffer),
           NO_CALLBACK,
           NULL,
           NO_FLAGS,
           &error);
if (error != SCI_ERR_OK) {
fprintf(stderr, "Client: SCIStartDmaTransfer failed - Error code 0x%x\n", error);
break;
}

// Wait for DMA transfer to complete
SCIWaitForDMAQueue(dma_queue, SCI_INFINITE_TIMEOUT, NO_FLAGS, &error);
if (error != SCI_ERR_OK) {
fprintf(stderr, "Client: SCIWaitForDMAQueue failed - Error code 0x%x\n", error);
break;
}

// Signal server that data is ready
SCIFlush(NULL, NO_FLAGS);
remote_seg->packet.cmd = CMD_DATA_READY;
SCIFlush(NULL, NO_FLAGS);

// Wait for server to echo back the frame number
local_seg->packet.cmd = CMD_INVALID;
while (local_seg->packet.cmd != CMD_DATA_READY) {
// Just wait
}

// Verify echoed frame number
int echoed_frame = *((int*)local_seg->message_buffer);
if (echoed_frame != numframes) {
fprintf(stderr, "Client: Server echoed wrong frame number %d (expected %d)\n", 
   echoed_frame, numframes);
}

// Process the frame
//c63_encode_image(cm, image);

// Clean up the image
free(image->Y);
free(image->U);
free(image->V);
free(image);

printf("Done!\n");

++numframes;

if (limit_numframes && numframes >= limit_numframes) {
break;
}
}

// Signal server to quit
remote_seg->packet.cmd = CMD_QUIT;
SCIFlush(NULL, NO_FLAGS);

printf("Client: Finished processing %d frames\n", numframes);
return numframes;
}

static void print_help()
{
    printf("Usage: ./c63client -r nodeid [options] input_file\n");
    printf("Commandline options:\n");
    printf("  -r                             Node id of server\n");
    printf("  -h                             Height of images to compress\n");
    printf("  -w                             Width of images to compress\n");
    printf("  -o                             Output file (.c63)\n");
    printf("  [-f]                           Limit number of frames to encode\n");
    printf("\n");

    exit(EXIT_FAILURE);
}

int main(int argc, char **argv)
{
    unsigned int localAdapterNo = 0;
    int c;
    yuv_t *image;
    sci_error_t error;
    
    if (argc == 1) {
        print_help();
    }

    while ((c = getopt(argc, argv, "r:h:w:o:f:i:")) != -1)
    {
        switch (c)
        {
            case 'r':
                remote_node = atoi(optarg);
                break;
            case 'h':
                height = atoi(optarg);
                break;
            case 'w':
                width = atoi(optarg);
                break;
            case 'o':
                output_file = optarg;
                break;
            case 'f':
                limit_numframes = atoi(optarg);
                break;
            default:
                print_help();
                break;
        }
    }

    if (optind >= argc)
    {
        fprintf(stderr, "Error getting program options, try --help.\n");
        exit(EXIT_FAILURE);
    }

    input_file = argv[optind];

    if (remote_node == 0) {
        fprintf(stderr, "Remote node-id is not specified. Use -r <remote node-id>\n");
        exit(EXIT_FAILURE);
    }

    // Open output file
    outfile = fopen(output_file, "wb");
    if (outfile == NULL)
    {
        perror("fopen");
        exit(EXIT_FAILURE);
    }

    // Initialize encoder
    struct c63_common *cm = init_c63_enc(width, height);
    cm->e_ctx.fp = outfile;

    if (limit_numframes)
    {
        printf("Limited to %d frames.\n", limit_numframes);
    }

    // Open input file
    FILE *infile = fopen(input_file, "rb");
    if (infile == NULL)
    {
        perror("fopen");
        exit(EXIT_FAILURE);
    }

    // Initialize SISCI
    SCIInitialize(NO_FLAGS, &error);
    if (error != SCI_ERR_OK) {
        fprintf(stderr, "SCIInitialize failed: %s\n", SCIGetErrorString(error));
        exit(EXIT_FAILURE);
    }

    // Set up SISCI resources
    sci_desc_t sd;
    sci_local_segment_t localSegment;
    sci_remote_segment_t remoteSegment;
    sci_map_t localMap, remoteMap;
    sci_dma_queue_t dmaQueue;
    volatile struct client_segment *client_segment;
    volatile struct server_segment *server_segment;

    // Open virtual device
    SCIOpen(&sd, NO_FLAGS, &error);
    if (error != SCI_ERR_OK) {
        fprintf(stderr, "SCIOpen failed - Error code 0x%x\n", error);
        SCITerminate();
        exit(EXIT_FAILURE);
    }
    
    // Create local segment
    SCICreateSegment(sd,
                     &localSegment,
                     SEGMENT_CLIENT,
                     sizeof(struct client_segment),
                     NO_CALLBACK,
                     NULL,
                     NO_FLAGS,
                     &error);
    if (error != SCI_ERR_OK) {
        fprintf(stderr, "SCICreateSegment failed - Error code 0x%x\n", error);
        SCIClose(sd, NO_FLAGS, &error);
        SCITerminate();
        exit(EXIT_FAILURE);
    }
    
    // Prepare segment
    SCIPrepareSegment(localSegment, localAdapterNo, NO_FLAGS, &error);
    if (error != SCI_ERR_OK) {
        fprintf(stderr, "SCIPrepareSegment failed - Error code 0x%x\n", error);
        SCIRemoveSegment(localSegment, NO_FLAGS, &error);
        SCIClose(sd, NO_FLAGS, &error);
        SCITerminate();
        exit(EXIT_FAILURE);
    }
    
    // Create DMA queue
    SCICreateDMAQueue(sd, &dmaQueue, localAdapterNo, 1, NO_FLAGS, &error);
    if (error != SCI_ERR_OK) {
        fprintf(stderr, "SCICreateDMAQueue failed - Error code 0x%x\n", error);
        SCIRemoveSegment(localSegment, NO_FLAGS, &error);
        SCIClose(sd, NO_FLAGS, &error);
        SCITerminate();
        exit(EXIT_FAILURE);
    }
    
    // Map local segment
    client_segment = (volatile struct client_segment *)SCIMapLocalSegment(
        localSegment, 
        &localMap, 
        0, 
        sizeof(struct client_segment), 
        NULL, 
        NO_FLAGS, 
        &error);
    
    if (error != SCI_ERR_OK) {
        fprintf(stderr, "SCIMapLocalSegment failed - Error code 0x%x\n", error);
        SCIRemoveDMAQueue(dmaQueue, NO_FLAGS, &error);
        SCIRemoveSegment(localSegment, NO_FLAGS, &error);
        SCIClose(sd, NO_FLAGS, &error);
        SCITerminate();
        exit(EXIT_FAILURE);
    }
    
    // Initialize control packet
    client_segment->packet.cmd = CMD_INVALID;
    
    // Make segment available
    SCISetSegmentAvailable(localSegment, localAdapterNo, NO_FLAGS, &error);
    if (error != SCI_ERR_OK) {
        fprintf(stderr, "SCISetSegmentAvailable failed - Error code 0x%x\n", error);
        SCIUnmapSegment(localMap, NO_FLAGS, &error);
        SCIRemoveDMAQueue(dmaQueue, NO_FLAGS, &error);
        SCIRemoveSegment(localSegment, NO_FLAGS, &error);
        SCIClose(sd, NO_FLAGS, &error);
        SCITerminate();
        exit(EXIT_FAILURE);
    }
    
    printf("Client: Connecting to server segment...\n");
    
    // Connect to server segment
    do {
        SCIConnectSegment(sd,
                          &remoteSegment,
                          remote_node,
                          SEGMENT_SERVER,
                          localAdapterNo,
                          NO_CALLBACK,
                          NULL,
                          SCI_INFINITE_TIMEOUT,
                          NO_FLAGS,
                          &error);
    } while (error != SCI_ERR_OK);
    
    printf("Client: Connected to server segment\n");
    
    // Map remote segment
    server_segment = (volatile struct server_segment *)SCIMapRemoteSegment(
        remoteSegment, 
        &remoteMap, 
        0,
        sizeof(struct server_segment),
        NULL, 
        NO_FLAGS, 
        &error);
    
    if (error != SCI_ERR_OK) {
        fprintf(stderr, "SCIMapRemoteSegment failed - Error code 0x%x\n", error);
        SCIDisconnectSegment(remoteSegment, NO_FLAGS, &error);
        SCISetSegmentUnavailable(localSegment, localAdapterNo, NO_FLAGS, &error);
        SCIUnmapSegment(localMap, NO_FLAGS, &error);
        SCIRemoveDMAQueue(dmaQueue, NO_FLAGS, &error);
        SCIRemoveSegment(localSegment, NO_FLAGS, &error);
        SCIClose(sd, NO_FLAGS, &error);
        SCITerminate();
        exit(EXIT_FAILURE);
    }
    
    // Enter main processing loop
    main_client_loop(cm, infile, limit_numframes, client_segment, server_segment, 
                     dmaQueue, localSegment, remoteSegment);
    
    // Clean up resources
    fclose(outfile);
    fclose(infile);
    free_c63_enc(cm);
    
    SCIDisconnectSegment(remoteSegment, NO_FLAGS, &error);
    SCIUnmapSegment(remoteMap, NO_FLAGS, &error);
    SCISetSegmentUnavailable(localSegment, localAdapterNo, NO_FLAGS, &error);
    SCIUnmapSegment(localMap, NO_FLAGS, &error);
    SCIRemoveDMAQueue(dmaQueue, NO_FLAGS, &error);
    SCIRemoveSegment(localSegment, NO_FLAGS, &error);
    SCIClose(sd, NO_FLAGS, &error);
    SCITerminate();
    
    return 0;
}